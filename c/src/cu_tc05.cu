#include "hip/hip_runtime.h"
#include "cu_tc05.hpp"

#include <chrono>
#include <hip/hip_runtime.h>
#include <>
#include <iomanip>
#include <iostream>

namespace cu::crypto::tc05
{
    static constexpr uint32_t GRDSZ = 1 << 16;
    static constexpr uint32_t BLKSZ = 512;
    static constexpr uint32_t VECSZ = 16;
    static constexpr size_t KNOWN_MSG_N = 16;

    __constant__ uint32_t known_msg_gpu[KNOWN_MSG_N];
    __constant__ uint32_t known_cip_gpu[KNOWN_MSG_N];
    __constant__ uint64_t base_key_gpu;
    __managed__ uint64_t guess_key_shr;

    namespace device
    {
        __device__ void static inline cmov(int c, uint64_t *x, uint64_t y)
        {
#if 1
            if (c)
                *x = y;
#else
            asm volatile inline("{\n\t"                       //
                                " .reg .pred %p;\n\t"         // declare p-register
                                " setp.eq.s32 %p, %1, 1;\n\t" // c == 1?
                                " @%p st.u64 [%0], %2;\n\t"   // set *x to y if true
                                "}"                           //
                                ::"l"(x),
                                "r"(c), "l"(y)
                                : "memory");
#endif
        }

        __device__ static inline uint16_t sigma(uint16_t word)
        {
            uint16_t new_word = 0;

            new_word |= (word & 0xC00C) >> 1;
            new_word |= (word & 0x0020) >> 2;
            new_word |= (word & 0x0010) >> 4;
            new_word |= (word & 0x0C00) >> 5;
            new_word |= (word & 0x2000) >> 6;
            new_word |= (word & 0x1000) >> 8;

            new_word |= (word & 0x00C0) << 3;
            new_word |= (word & 0x0100) << 4;
            new_word |= (word & 0x0200) << 6;
            new_word |= (word & 0x0001) << 8;
            new_word |= (word & 0x0002) << 10;

            return new_word;
        }

        __device__ static inline uint32_t sigma_par(uint32_t word)
        {
            uint32_t new_word = 0;

            new_word |= (word & 0xC00CC00C) >> 1;
            new_word |= (word & 0x00200020) >> 2;
            new_word |= (word & 0x00100010) >> 4;
            new_word |= (word & 0x0C000C00) >> 5;
            new_word |= (word & 0x20002000) >> 6;
            new_word |= (word & 0x10001000) >> 8;

            new_word |= (word & 0x00C000C0) << 3;
            new_word |= (word & 0x01000100) << 4;
            new_word |= (word & 0x02000200) << 6;
            new_word |= (word & 0x00010001) << 8;
            new_word |= (word & 0x00020002) << 10;

            return new_word;
        }

        __device__ static inline uchar4 sigma_v2(uchar4 m)
        {
            uchar4 c;

            c.x = (m.y & 2) << 2;
            c.x |= m.y & 1;
            c.x |= (m.x >> 1) & 6;

            c.y = (m.w & 2) << 2;
            c.y |= m.w & 1;
            c.y |= (m.z >> 1) & 6;

            c.z = (m.x & 2) << 2;
            c.z |= m.x & 1;
            c.z |= (m.y >> 1) & 6;

            c.w = (m.z & 2) << 2;
            c.w |= m.z & 1;
            c.w |= (m.w >> 1) & 6;

            return c;
        }

        __device__ static inline uint16_t sbox(uint16_t m)
        {
            static constexpr uint16_t sbox[16] = {0xE, 0xB, 0x4, 0x6, 0xA, 0xD, 0x7, 0x0,
                                                  0x3, 0x8, 0xF, 0xC, 0x5, 0x9, 0x1, 0x2};
            uint16_t p = sbox[m & 0xF];

            p |= sbox[m >> 4 & 0xF] << 4;
            p |= sbox[m >> 8 & 0xF] << 8;
            p |= sbox[m >> 12] << 12;

            return p;
        }

        __device__ static inline uint32_t sbox_par(uint32_t m)
        {
            static constexpr uint16_t sbox[16] = {0xE, 0xB, 0x4, 0x6, 0xA, 0xD, 0x7, 0x0,
                                                  0x3, 0x8, 0xF, 0xC, 0x5, 0x9, 0x1, 0x2};
            uint32_t p = sbox[m & 0xF];

            p |= sbox[m >> 4 & 0xF] << 4;
            p |= sbox[m >> 8 & 0xF] << 8;
            p |= sbox[m >> 12 & 0xF] << 12;
            p |= sbox[m >> 16 & 0xF] << 16;
            p |= sbox[m >> 20 & 0xF] << 20;
            p |= sbox[m >> 24 & 0xF] << 24;
            p |= sbox[m >> 28 & 0xF] << 28;

            return p;
        }

        template<uint32_t rounds>
        __device__ static inline uint32_t enc(uint32_t m, uint64_t k)
        {
            uint16_t sk[4] = {static_cast<uint16_t>(k >> 48), static_cast<uint16_t>(k >> 32),
                              static_cast<uint16_t>(k >> 16), static_cast<uint16_t>(k)};
            uint16_t l = m >> 16;
            uint16_t r = m;

#pragma unroll(rounds)
            for (uint32_t i = 0; i < rounds; ++i)
            {
                uint16_t t = l;
                l = sigma(sbox(l)) ^ r ^ sk[i & 3];
                r = t;

                sk[i & 3] ^= sk[(i - 1) & 3];
                sk[i & 3] ^= sigma(sk[(i - 2) & 3]);
                sk[i & 3] ^= 0xC;
            }

            return (uint32_t)l << 16 | r;
        }

        template<uint32_t rounds>
        __device__ static inline uint32_t dec(uint32_t m, uint64_t last_key)
        {
            uint16_t sk[4];
            uint16_t l = m >> 16;
            uint16_t r = m;

            sk[(rounds - 1) & 3] = last_key >> 48;
            sk[(rounds - 2) & 3] = last_key >> 32;
            sk[(rounds - 3) & 3] = last_key >> 16;
            sk[(rounds - 4) & 3] = last_key >> 0;


#pragma unroll
            for (uint32_t i = rounds - 1; i >= 4; --i)
            {
                uint16_t t = r;
                r = sbox(r);
                r = sigma(r);
                r ^= l;
                r ^= sk[i & 3];
                l = t;

                sk[i & 3] ^= sk[(i - 1) & 3];
                sk[i & 3] ^= sigma(sk[(i - 2) & 3]);
                sk[i & 3] ^= 0xC;
            }

#pragma unroll
            for (uint32_t i = 3; i != ~0U; --i)
            {
                uint16_t t = r;
                r = sbox(r);
                r = sigma(r);
                r ^= l;
                r ^= sk[i & 3];
                l = t;
            }

            return (uint32_t)l << 16 | r;
        }

        template<uint32_t rounds>
        __device__ static inline uint64_t dec_par(uint32_t m1, uint32_t m2, uint64_t last_key)
        {
            uint32_t sk[4];
            uint32_t l = (m1 & 0xFFFF0000) | (m2 >> 16);
            uint32_t r = (m1 << 16) | (m2 & 0x0000FFFF);
            uint32_t t;

            sk[(rounds - 1) & 3] = (last_key >> 48) & 0xFFFF;
            sk[(rounds - 2) & 3] = (last_key >> 32) & 0xFFFF;
            sk[(rounds - 3) & 3] = (last_key >> 16) & 0xFFFF;
            sk[(rounds - 4) & 3] = (last_key >> 0) & 0xFFFF;

            sk[0] |= sk[0] << 16;
            sk[1] |= sk[1] << 16;
            sk[2] |= sk[2] << 16;
            sk[3] |= sk[3] << 16;

#pragma unroll
            for (uint32_t i = rounds - 1; i >= 4; --i)
            {
                t = r;
                r = sbox_par(r);
                r = sigma_par(r);
                r ^= l;
                r ^= sk[i & 3];
                l = t;

                sk[i & 3] ^= sk[(i - 1) & 3];
                sk[i & 3] ^= sigma_par(sk[(i - 2) & 3]);
                sk[i & 3] ^= 0x000C000C;
            }

#pragma unroll
            for (uint32_t i = 3; i != ~0U; --i)
            {
                t = r;
                r = sbox_par(r);
                r = sigma_par(r);
                r ^= l;
                r ^= sk[i & 3];
                l = t;
            }

            m1 = (l & 0xFFFF0000) | (r >> 16);
            m2 = (l << 16) | (r & 0x0000FFFF);

            return (uint64_t)m1 << 32 | m2;
        }

        __device__ static inline uint32_t dec8(uint32_t m, uint64_t last_key)
        {
            uint16_t sk[4];
            uint16_t t;
            uint16_t l = m >> 16;
            uint16_t r = m;

            sk[3] = last_key >> 48;
            sk[2] = last_key >> 32;
            sk[1] = last_key >> 16;
            sk[0] = last_key >> 0;

            t = r;
            r = sbox(r);
            r = sigma(r);
            r ^= l;
            r ^= sk[3];
            l = t;
            sk[3] ^= sk[2];
            sk[3] ^= sigma(sk[1]);
            sk[3] ^= 0xC;

            t = r;
            r = sbox(r);
            r = sigma(r);
            r ^= l;
            r ^= sk[2];
            l = t;
            sk[2] ^= sk[1];
            sk[2] ^= sigma(sk[0]);
            sk[2] ^= 0xC;

            t = r;
            r = sbox(r);
            r = sigma(r);
            r ^= l;
            r ^= sk[1];
            l = t;
            sk[1] ^= sk[0];
            sk[1] ^= sigma(sk[3]);
            sk[1] ^= 0xC;

            t = r;
            r = sbox(r);
            r = sigma(r);
            r ^= l;
            r ^= sk[0];
            l = t;
            sk[0] ^= sk[3];
            sk[0] ^= sigma(sk[2]);
            sk[0] ^= 0xC;

            t = r;
            r = sbox(r);
            r = sigma(r);
            r ^= l;
            r ^= sk[3];
            l = t;

            t = r;
            r = sbox(r);
            r = sigma(r);
            r ^= l;
            r ^= sk[2];
            l = t;

            t = r;
            r = sbox(r);
            r = sigma(r);
            r ^= l;
            r ^= sk[1];
            l = t;

            t = r;
            r = sbox(r);
            r = sigma(r);
            r ^= l;
            r ^= sk[0];

            return ((uint32_t)t << 16) | r;
        }

        __device__ static inline uint32_t dec8_v2(uint32_t m, uint64_t last_key)
        {
            static constexpr uint8_t sbox[16] = {0xE, 0xB, 0x4, 0x6, 0xA, 0xD, 0x7, 0x0,
                                                 0x3, 0x8, 0xF, 0xC, 0x5, 0x9, 0x1, 0x2};

            uint32_t hk = last_key >> 32;
            uint32_t lk = last_key;

            uchar4 sk[4] = {
                {
                    static_cast<uint8_t>((lk >> 12) & 0xF),
                    static_cast<uint8_t>((lk >> 8) & 0xF),
                    static_cast<uint8_t>((lk >> 4) & 0xF),
                    static_cast<uint8_t>((lk >> 0) & 0xF),
                },
                {
                    static_cast<uint8_t>((lk >> 28) & 0xF),
                    static_cast<uint8_t>((lk >> 24) & 0xF),
                    static_cast<uint8_t>((lk >> 20) & 0xF),
                    static_cast<uint8_t>((lk >> 16) & 0xF),
                },
                {
                    static_cast<uint8_t>((hk >> 12) & 0xF),
                    static_cast<uint8_t>((hk >> 8) & 0xF),
                    static_cast<uint8_t>((hk >> 4) & 0xF),
                    static_cast<uint8_t>((hk >> 0) & 0xF),
                },
                {
                    static_cast<uint8_t>((hk >> 28) & 0xF),
                    static_cast<uint8_t>((hk >> 24) & 0xF),
                    static_cast<uint8_t>((hk >> 20) & 0xF),
                    static_cast<uint8_t>((hk >> 16) & 0xF),
                },
            };

            uchar4 t;

            uchar4 l = {
                static_cast<uint8_t>((m >> 28) & 0xF),
                static_cast<uint8_t>((m >> 24) & 0xF),
                static_cast<uint8_t>((m >> 20) & 0xF),
                static_cast<uint8_t>((m >> 16) & 0xF),
            };

            uchar4 r = {
                static_cast<uint8_t>((m >> 12) & 0xF),
                static_cast<uint8_t>((m >> 8) & 0xF),
                static_cast<uint8_t>((m >> 4) & 0xF),
                static_cast<uint8_t>((m >> 0) & 0xF),
            };

            {
                t = r;
                r.x = sbox[r.x];
                r.y = sbox[r.y];
                r.z = sbox[r.z];
                r.w = sbox[r.w];
                r = sigma_v2(r);
                r.x ^= l.x;
                r.y ^= l.y;
                r.z ^= l.z;
                r.w ^= l.w;
                r.x ^= sk[3].x;
                r.y ^= sk[3].y;
                r.z ^= sk[3].z;
                r.w ^= sk[3].w;
                l = t;

                sk[3].x ^= sk[2].x;
                sk[3].y ^= sk[2].y;
                sk[3].z ^= sk[2].z;
                sk[3].w ^= sk[2].w;
                t = sigma_v2(sk[1]);
                sk[3].x ^= t.x;
                sk[3].y ^= t.y;
                sk[3].z ^= t.z;
                sk[3].w ^= t.w;
                sk[3].w ^= 0xC;
            }

            {
                t = r;
                r.x = sbox[r.x];
                r.y = sbox[r.y];
                r.z = sbox[r.z];
                r.w = sbox[r.w];
                r = sigma_v2(r);
                r.x ^= l.x;
                r.y ^= l.y;
                r.z ^= l.z;
                r.w ^= l.w;
                r.x ^= sk[2].x;
                r.y ^= sk[2].y;
                r.z ^= sk[2].z;
                r.w ^= sk[2].w;
                l = t;

                sk[2].x ^= sk[1].x;
                sk[2].y ^= sk[1].y;
                sk[2].z ^= sk[1].z;
                sk[2].w ^= sk[1].w;
                t = sigma_v2(sk[0]);
                sk[2].x ^= t.x;
                sk[2].y ^= t.y;
                sk[2].z ^= t.z;
                sk[2].w ^= t.w;
                sk[2].w ^= 0xC;
            }

            {
                t = r;
                r.x = sbox[r.x];
                r.y = sbox[r.y];
                r.z = sbox[r.z];
                r.w = sbox[r.w];
                r = sigma_v2(r);
                r.x ^= l.x;
                r.y ^= l.y;
                r.z ^= l.z;
                r.w ^= l.w;
                r.x ^= sk[1].x;
                r.y ^= sk[1].y;
                r.z ^= sk[1].z;
                r.w ^= sk[1].w;
                l = t;

                sk[1].x ^= sk[0].x;
                sk[1].y ^= sk[0].y;
                sk[1].z ^= sk[0].z;
                sk[1].w ^= sk[0].w;
                t = sigma_v2(sk[3]);
                sk[1].x ^= t.x;
                sk[1].y ^= t.y;
                sk[1].z ^= t.z;
                sk[1].w ^= t.w;
                sk[1].w ^= 0xC;
            }

            {
                t = r;
                r.x = sbox[r.x];
                r.y = sbox[r.y];
                r.z = sbox[r.z];
                r.w = sbox[r.w];
                r = sigma_v2(r);
                r.x ^= l.x;
                r.y ^= l.y;
                r.z ^= l.z;
                r.w ^= l.w;
                r.x ^= sk[0].x;
                r.y ^= sk[0].y;
                r.z ^= sk[0].z;
                r.w ^= sk[0].w;
                l = t;

                sk[0].x ^= sk[3].x;
                sk[0].y ^= sk[3].y;
                sk[0].z ^= sk[3].z;
                sk[0].w ^= sk[3].w;
                t = sigma_v2(sk[2]);
                sk[0].x ^= t.x;
                sk[0].y ^= t.y;
                sk[0].z ^= t.z;
                sk[0].w ^= t.w;
                sk[0].w ^= 0xC;
            }

            {
                t = r;
                r.x = sbox[r.x];
                r.y = sbox[r.y];
                r.z = sbox[r.z];
                r.w = sbox[r.w];
                r = sigma_v2(r);
                r.x ^= l.x;
                r.y ^= l.y;
                r.z ^= l.z;
                r.w ^= l.w;
                r.x ^= sk[3].x;
                r.y ^= sk[3].y;
                r.z ^= sk[3].z;
                r.w ^= sk[3].w;
                l = t;
            }

            {
                t = r;
                r.x = sbox[r.x];
                r.y = sbox[r.y];
                r.z = sbox[r.z];
                r.w = sbox[r.w];
                r = sigma_v2(r);
                r.x ^= l.x;
                r.y ^= l.y;
                r.z ^= l.z;
                r.w ^= l.w;
                r.x ^= sk[2].x;
                r.y ^= sk[2].y;
                r.z ^= sk[2].z;
                r.w ^= sk[2].w;
                l = t;
            }

            {
                t = r;
                r.x = sbox[r.x];
                r.y = sbox[r.y];
                r.z = sbox[r.z];
                r.w = sbox[r.w];
                r = sigma_v2(r);
                r.x ^= l.x;
                r.y ^= l.y;
                r.z ^= l.z;
                r.w ^= l.w;
                r.x ^= sk[1].x;
                r.y ^= sk[1].y;
                r.z ^= sk[1].z;
                r.w ^= sk[1].w;
                l = t;
            }

            {
                t = r;
                r.x = sbox[r.x];
                r.y = sbox[r.y];
                r.z = sbox[r.z];
                r.w = sbox[r.w];
                r = sigma_v2(r);
                r.x ^= l.x;
                r.y ^= l.y;
                r.z ^= l.z;
                r.w ^= l.w;
                r.x ^= sk[0].x;
                r.y ^= sk[0].y;
                r.z ^= sk[0].z;
                r.w ^= sk[0].w;
            }

            uint32_t c = r.w;

            c |= static_cast<uint32_t>(r.z) << 4;
            c |= static_cast<uint32_t>(r.y) << 8;
            c |= static_cast<uint32_t>(r.x) << 12;
            c |= static_cast<uint32_t>(t.w) << 16;
            c |= static_cast<uint32_t>(t.z) << 20;
            c |= static_cast<uint32_t>(t.y) << 24;
            c |= static_cast<uint32_t>(t.x) << 28;

            return c;
        }

        template<uint32_t rounds>
        __global__ void test_enc(uint32_t msg, uint64_t key, uint32_t *cip)
        {
            *cip = enc<rounds>(msg, key);
        }

        template<uint32_t rounds>
        __global__ void test_dec(uint32_t cip, uint64_t last_key, uint32_t *msg)
        {
            *msg = dec<rounds>(cip, last_key);
        }

        template<uint32_t rounds>
        __global__ void crack_enc(uint64_t off)
        {
            // concatenation of the 4 last subkeys, must be de-scheduled
            uint64_t key = base_key_gpu;
            uint32_t id = (blockIdx.x * blockDim.x + threadIdx.x) * VECSZ;
            //uint64_t m12 = (uint64_t)known_msg_gpu[0] << 32 | known_msg_gpu[1];

            off += id;

#pragma unroll
            for (uint32_t i = 0; i < VECSZ; ++i)
            {
                key &= 0xFFFFFFFF00000000;
                key |= (off + i);

                bool flag = dec<rounds>(known_cip_gpu[0], key) == known_msg_gpu[0];

                if (flag) [[unlikely]]
                {
                    flag = dec<rounds>(known_cip_gpu[1], key) == known_msg_gpu[1];
                    if (flag) [[unlikely]]
                        guess_key_shr = key;
                }
            }
        }

        __global__ void crack_enc8(uint64_t off)
        {
            // concatenation of the 4 last subkeys, must be de-scheduled
            bool flag;
            uint64_t key = base_key_gpu;

            off += (blockIdx.x * blockDim.x + threadIdx.x) * VECSZ;
            key |= off;

            flag = (dec8(known_cip_gpu[0], key) == known_msg_gpu[0]);
            if (flag)
            {
                flag &= dec8(known_cip_gpu[1], key) == known_msg_gpu[1];
                if (flag)
                    guess_key_shr = key;
            }

            ++key;
            flag = (dec8(known_cip_gpu[0], key) == known_msg_gpu[0]);
            if (flag)
            {
                flag &= dec8(known_cip_gpu[1], key) == known_msg_gpu[1];
                if (flag)
                    guess_key_shr = key;
            }

            ++key;
            flag = (dec8(known_cip_gpu[0], key) == known_msg_gpu[0]);
            if (flag)
            {
                flag &= dec8(known_cip_gpu[1], key) == known_msg_gpu[1];
                if (flag)
                    guess_key_shr = key;
            }

            ++key;
            flag = (dec8(known_cip_gpu[0], key) == known_msg_gpu[0]);
            if (flag)
            {
                flag &= dec8(known_cip_gpu[1], key) == known_msg_gpu[1];
                if (flag)
                    guess_key_shr = key;
            }
        }
    } // namespace device

    template<uint32_t rounds>
    uint32_t test_enc(uint32_t msg, uint64_t key)
    {
        uint32_t cip;
        uint32_t *d_cip;

        hipMalloc(&d_cip, sizeof(*d_cip));
        device::test_enc<rounds><<<1, 1>>>(msg, key, d_cip);
        hipDeviceSynchronize();
        hipMemcpy(&cip, d_cip, sizeof(cip), hipMemcpyDeviceToHost);
        hipFree(d_cip);

        return cip;
    }

    uint32_t test_enc(uint32_t msg, uint64_t key)
    {
        return test_enc<ROUNDS_N>(msg, key);
    }

    template<uint32_t rounds>
    uint32_t test_dec(uint32_t cip, uint64_t last_key)
    {
        uint32_t msg;
        uint32_t *d_msg;

        hipMalloc(&d_msg, sizeof(*d_msg));
        device::test_dec<rounds><<<1, 1>>>(cip, last_key, d_msg);
        hipDeviceSynchronize();
        hipMemcpy(&msg, d_msg, sizeof(msg), hipMemcpyDeviceToHost);
        hipFree(d_msg);

        return msg;
    }

    uint32_t test_dec(uint32_t cip, uint64_t last_key)
    {
        return test_dec<ROUNDS_N>(cip, last_key);
    }

    template<uint32_t rounds>
    uint64_t crack(std::span<uint32_t> msg, std::span<uint32_t> cip, uint16_t skn1, uint16_t skn2,
                   uint64_t off, size_t watch)
    {
        using namespace std::chrono;
        using clk = high_resolution_clock;
        static constexpr uint64_t RECOVER_SPACE = 1ULL << 32;

        if (test_enc<rounds>(msg[0], 0) == cip[0])
            return 0;

        hipStream_t compute_stream;

        hipStreamCreate(&compute_stream);

        hipMemcpyToSymbol(HIP_SYMBOL(known_msg_gpu), msg.data(), KNOWN_MSG_N * sizeof(*known_msg_gpu));
        hipMemcpyToSymbol(HIP_SYMBOL(known_cip_gpu), cip.data(), KNOWN_MSG_N * sizeof(*known_cip_gpu));
        uint64_t base_key = (uint64_t)skn1 << 48 | (uint64_t)skn2 << 32;
        hipMemcpyToSymbol(HIP_SYMBOL(base_key_gpu), &base_key, sizeof(base_key));

        std::cout << std::left;

        auto start = clk::now();
        for (size_t i = 0; !guess_key_shr && off < RECOVER_SPACE; ++i, off += GRDSZ * BLKSZ * VECSZ)
        {
            device::crack_enc<rounds><<<GRDSZ, BLKSZ, 0, compute_stream>>>(off);
            if (watch && !(i % watch))
            {
                auto elap = clk::now() - start;
                auto elap_ns = duration_cast<nanoseconds>(elap).count();
                auto elap_sec = duration_cast<seconds>(elap).count();

                std::cout << std::setw(20) << off << '\t' << elap_sec << " sec. "
                          << "(" << ((double)off / elap_sec) << " enc/s)\r";
                std::cout.flush();
            }
        }
        if (watch)
            std::cout << '\n';

        std::cout << std::right;
        hipDeviceSynchronize();
        hipStreamDestroy(compute_stream);

        uint64_t key = guess_key_shr;

        guess_key_shr = 0;

        if (key)
        {
            uint16_t sk[4];

            sk[(rounds - 1) & 3] = key >> 48;
            sk[(rounds - 2) & 3] = key >> 32;
            sk[(rounds - 3) & 3] = key >> 16;
            sk[(rounds - 4) & 3] = key >> 0;

            for (uint32_t i = rounds - 1; i >= 4; --i)
                ::crypto::tc05::next_key(sk, i);

            key = sk[3];
            key |= static_cast<uint64_t>(sk[2]) << 16;
            key |= static_cast<uint64_t>(sk[1]) << 32;
            key |= static_cast<uint64_t>(sk[0]) << 48;
        }

        return key;
    }

    uint64_t crack(std::span<uint32_t> msg, std::span<uint32_t> cip, uint16_t skn2, uint16_t skn1,
                   uint64_t off, size_t watch)
    {
        return crack<ROUNDS_N>(msg, cip, skn2, skn1, off, watch);
    }

} // namespace cu::crypto::tc05
